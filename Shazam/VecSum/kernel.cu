// System includes
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hipfft/hipfft.h>
// CUDA runtime
#include <hip/hip_runtime.h>


#include <math.h>

// Adds an additional library so that timeGetTime() can be used

#include <stdlib.h>

#include <time.h>
#include <omp.h>

/**
* Matrix multiplication (CUDA Kernel) on the device: C = A * B
*/

#define NX 256
#define BATCH 1

__global__ void
compareKernel(int *A, int *B, int *C) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	//int Pvalue = 0;
	C[i] = abs(A[i] - B[i]);

		
}




int readFile(int **grades, char *addr);








int CompareWav()
{
	
	int *h_A = NULL;
	unsigned int count_A;
	count_A = readFile(&h_A, "M1.txt");
	
	
	int *h_B = NULL;
	unsigned int count_B;
	count_B = readFile(&h_B, "M2.txt");


	unsigned int size_A = sizeof(int)* count_A;
	unsigned int size_B = sizeof(int)* count_B;

	unsigned int MinCount= count_B;

	if (count_A < count_B) {
		MinCount = count_A;
	}

	unsigned int MaxCount = count_B;

	if (count_A > count_B) {
		MaxCount = count_A;
	}



	// Allocate device memory
	int *d_A, *d_B, *d_C;

	// Allocate host matrix C
	unsigned int size_C = sizeof(int)* MinCount;
	int *h_C = (int *)malloc(size_C);

	if (h_C == NULL)
	{
		fprintf(stderr, "Failed to allocate host matrix C!\n");
		exit(EXIT_FAILURE);
	}

	hipError_t error;

	error = hipMalloc((void **)&d_A, size_A);

	if (error != hipSuccess)
	{
		printf("hipMalloc d_A returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void **)&d_B, size_B);

	if (error != hipSuccess)
	{
		printf("hipMalloc d_B returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void **)&d_C, size_C);

	if (error != hipSuccess)
	{
		printf("hipMalloc d_C returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	// copy host memory to device
	error = hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);

	if (error != hipSuccess)
	{
		printf("hipMemcpy (d_A,h_A) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

	if (error != hipSuccess)
	{
		printf("hipMemcpy (d_B,h_B) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	// -------------cuFFT IS HERE ------------------
	
	hipfftHandle plan;
	hipfftComplex *data;
	/*
	cudaMalloc((void**)&data, sizeof(cufftComplex)*(NX / 2 + 1)*BATCH);
	if (cudaGetLastError() != cudaSuccess) {
		fprintf(stderr, "Cuda error: Failed to allocate\n");
		return;
	}
	*/
	if (hipfftPlan1d(&plan, NX, HIPFFT_R2C, BATCH) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: Plan creation failed");
		exit(EXIT_FAILURE);
	}
	
		
	if (hipfftExecR2C(plan, (hipfftReal*)d_A, (hipfftComplex*)d_A) != HIPFFT_SUCCESS) {
			fprintf(stderr, "CUFFT error: ExecC2C Forward failed");
			exit(EXIT_FAILURE);
		}
	if (hipDeviceSynchronize() != hipSuccess) {
		fprintf(stderr, "Cuda error: Failed to synchronize\n");
		exit(EXIT_FAILURE);
	}
	
	printf("cuFFT Done :)");

	int gridCount = ceil(MinCount / 1024);
	// Setup execution parameters
	dim3 threads(1024, 1, 1);
	dim3 grid(gridCount, 1, 1);

	
	// Create and start timer
	printf("Computing result using CUDA Kernel...\n");

	// Allocate CUDA events that we'll use for timing
	hipEvent_t start;
	error = hipEventCreate(&start);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	hipEvent_t stop;
	error = hipEventCreate(&stop);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Record the start event
	error = hipEventRecord(start, NULL);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}


	// Execute the kernel
	//compareKernel << < grid, threads >> > (d_A, d_B, d_C);
	//cudaDeviceSynchronize();
	error = hipGetLastError();
	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to launch kernel!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Record the stop event
	error = hipEventRecord(stop, NULL);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Wait for the stop event to complete
	error = hipEventSynchronize(stop);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	float msecTotal = 0.0f;
	error = hipEventElapsedTime(&msecTotal, start, stop);

	printf("Elapsed time in msec = %f\n", msecTotal);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Copy result from device to host
	error = hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

	if (error != hipSuccess)
	{
		printf("hipMemcpy (h_C,d_C) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}


	// Clean up memory
	free(h_A);
	free(h_B);
	free(h_C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);


	return EXIT_SUCCESS;

}



int readFile(int **grades, char *addr) {
	FILE *fp;
	int temp;
	//grades = NULL;
	int count = 1;
	long index;

	


	fp = fopen(addr, "rb+");

	while (fscanf(fp, "%d", &temp) != EOF)

	{


		if (*grades == NULL)

		{

			*grades = (int *)malloc(sizeof(temp));
			**grades = temp;

			printf("The grade is %d\r\n", temp);
		}

		else
		{
			//printf("The grade is realloc %d\r\n", temp);
			count++;
			*grades = (int *)realloc(*grades, sizeof(int)*count);
			index = count - 1;
			(*grades)[index] = temp;
			//printf("the index is %d\r\n",index);

		}

	}
	printf("Done Total %d numbers \n", count);
	fclose(fp);
	temp = 0;
	/*
	while (index >= 0)
	{

	printf("the read value is %d\r\n", (*grades)[temp]);
	index--;
	temp++;

	} */

	return(count);
	
}


/**
* Program main
*/
int main(int argc, char **argv)
{
	CompareWav();
	/*
	char *addr = argv[1];
	 int *grades=NULL;
	 unsigned int size;
	size= readFile(&grades,"M2.txt");
	int temp = 0;

	printf("Size is %d \n", &size);

	//temp = 0;
	
	while (size >= 0)
	{
		printf("Size is %d \n", size);
		printf("the read value is %d\r\n", grades[temp]);
		size--;
		temp++;

	}

	free(grades);
	*/


	/*

	// By default, we use device 0
	int devID = 0;
	cudaSetDevice(devID);

	cudaError_t error;
	cudaDeviceProp deviceProp;
	error = cudaGetDevice(&devID);

	if (error != cudaSuccess)
	{
		printf("cudaGetDevice returned error %s (code %d), line(%d)\n", cudaGetErrorString(error), error, __LINE__);
	}

	error = cudaGetDeviceProperties(&deviceProp, devID);

	if (deviceProp.computeMode == cudaComputeModeProhibited)
	{
		fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::cudaSetDevice().\n");
		exit(EXIT_SUCCESS);
	}

	if (error != cudaSuccess)
	{
		printf("cudaGetDeviceProperties returned error %s (code %d), line(%d)\n", cudaGetErrorString(error), error, __LINE__);
	}
	else
	{
		printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
	}
	size_t Mask_Width = M_Size;
	size_t Width = 100000000;
	//float* N=(float *)malloc(sizeof(float) * Width);
	//float* M = (float *)malloc(sizeof(float) * Mask_Width);
	//float* P = (float *)malloc(sizeof(float) * Width);
	// Size of square matrices

	//printf("[-] N = ");
	//scanf("%u", &n);

	

	exit(matrix_result);
	*/
}
