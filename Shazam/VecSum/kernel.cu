#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hipfft/hipfft.h>
// CUDA runtime
#include <hip/hip_runtime.h>
#include <>

#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
//#include <dirent.h>
// Adds an additional library so that timeGetTime() can be used

#include <stdlib.h>

#include <time.h>
#include <omp.h>



typedef float2 Complex;
/**
* Matrix multiplication (CUDA Kernel) on the device: C = A * B
*/

#define NX 256
#define BATCH 1



static __global__ void ComplexPointwiseMulAndScale(Complex* A, Complex* B, double* res, int sizeA, int sizeB)
{
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	double valueA = 0.0;
	double valueB = 0.0;
	double diff = 99999999999999999999999.9;
	//double fab[sizeA];
	
	//int i = threadID;

	for (int i = threadID; i < ((sizeA- sizeB)+1); i += numThreads) {
		if ((i + (sizeB - 1)) < sizeA) {
			res[i] = 0.0;
			for (int j = 0; j < sizeB; j++) {
					//valueA = sqrt(pow(A[i + j].x, 2) + pow(A[i + j].y, 2));
					//valueB = sqrt(pow(B[i].x, 2) + pow(B[i].y, 2));
					res[i] += fabs(sqrt(pow(A[i + j].x, 2) + pow(A[i + j].y, 2)) - sqrt(pow(B[i].x, 2) + pow(B[i].y, 2)));
					
				
			}
			if (res[i] < diff) {
				diff = res[i];
			}

		}
		res[(sizeA - sizeB) + 1] = diff;
	}
	//res[1] = fab;
}

int PadData(const Complex* signal, Complex** padded_signal, int signal_size,
	const Complex* filter_kernel, Complex** padded_filter_kernel, int filter_kernel_size)
{
	int minRadius = filter_kernel_size / 2;
	int maxRadius = filter_kernel_size - minRadius;
	int new_size = signal_size + maxRadius;

	// Pad signal
	Complex* new_data = (Complex*)malloc(sizeof(Complex) * new_size);
	memcpy(new_data + 0, signal, signal_size * sizeof(Complex));
	memset(new_data + signal_size, 0, (new_size - signal_size) * sizeof(Complex));
	*padded_signal = new_data;

	// Pad filter
	new_data = (Complex*)malloc(sizeof(Complex) * new_size);
	memcpy(new_data + 0, filter_kernel + minRadius, maxRadius * sizeof(Complex));
	memset(new_data + maxRadius, 0, (new_size - filter_kernel_size) * sizeof(Complex));
	memcpy(new_data + new_size - minRadius, filter_kernel, minRadius * sizeof(Complex));
	*padded_filter_kernel = new_data;

	return new_size;
}




int readFile(int **grades, char *addr);







int CompareWav(char *path1, char *path2, double *a)
{
	printf("%s --- %s --- \n", path1, path2);
	
	printf("[simpleCUFFT] is starting...\n");

	int *h_A_real = NULL;
	unsigned int count_A;
	count_A = readFile(&h_A_real, path1);


	int *h_B_real = NULL;
	unsigned int count_B;
	count_B = readFile(&h_B_real, path2);


	unsigned int count_C= (count_A - count_B) + 2;

	Complex* h_A = (Complex*)malloc(sizeof(Complex) * count_A);
	// Initalize the memory for the signal
	for (unsigned int i = 0; i < count_A; ++i) {
		printf("Int is %d \n", h_A_real[i]);
		h_A[i].x = h_A_real[i] + 0.0;
		printf("Num is %f \n", h_A[i].x);
		h_A[i].y = 0;
	}

	Complex* h_B = (Complex*)malloc(sizeof(Complex) * count_B);
	// Initalize the memory for the signal
	for (unsigned int i = 0; i < count_B; ++i) {
		printf("Int is %d \n", h_B_real[i]);
		h_B[i].x = h_B_real[i] + +0.0;;
		printf("Num is %f \n", h_B[i].x);
		h_B[i].y = 0;
	}

	/*

	// Pad signal and filter kernel
	Complex* h_padded_signal;
	Complex* h_padded_filter_kernel;
	int new_size = PadData(h_signal, &h_padded_signal, SIGNAL_SIZE,
		h_filter_kernel, &h_padded_filter_kernel, FILTER_KERNEL_SIZE);
	int mem_size = sizeof(Complex) * new_size;

	*/

	int MinCount = count_B;

	if (count_A < count_B) {
		MinCount = count_A;
	}

	unsigned int MaxCount = count_B;

	if (count_A > count_B) {
		MaxCount = count_A;
	}

	unsigned int size_A = sizeof(Complex)* count_A;
	unsigned int size_B = sizeof(Complex)* count_B;
	unsigned int size_C = sizeof(double)* count_C;
	// Allocate device memory for signal
	Complex* d_A;
	hipMalloc((void**)&d_A, size_A);
	// Copy host memory to device
	hipMemcpy(d_A, h_A, size_A,
		hipMemcpyHostToDevice);

	// Allocate device memory for filter kernel
	Complex* d_B;
	hipMalloc((void**)&d_B, size_B);

	// Copy host memory to device
	hipMemcpy(d_B, h_B, size_B,
		hipMemcpyHostToDevice);

	double *d_C;
	hipMalloc((void**)&d_C, count_C * sizeof(double));
	double *h_C = (double *)malloc(count_C * sizeof(double));

	// CUFFT plan
	hipfftHandle planA, planB;
	hipfftPlan1d(&planA, count_A, HIPFFT_C2C, 1);
	hipfftPlan1d(&planB, count_B, HIPFFT_C2C, 1);

	// Transform signal and kernel
	printf("Transforming signal hipfftExecC2C\n");
	hipfftExecC2C(planA, (hipfftComplex *)d_A, (hipfftComplex *)d_A, HIPFFT_FORWARD);
	hipfftExecC2C(planB, (hipfftComplex *)d_B, (hipfftComplex *)d_B, HIPFFT_FORWARD);

	// Multiply the coefficients together and normalize the result
	printf("Launching ComplexPointwiseMulAndScale<<< >>>\n");

	ComplexPointwiseMulAndScale << <32, 256 >> >(d_A, d_B, d_C, count_A,count_B);

	// Transform signal back
	printf("Transforming signal back hipfftExecC2C\n");
	//hipfftExecC2C(plan, (hipfftComplex *)d_A, (hipfftComplex *)d_A, HIPFFT_BACKWARD);

	// Copy device memory to host
	int Min_size = sizeof(Complex) * count_A;
	Complex* h_convolved_signal = (Complex*)malloc(sizeof(Complex) * count_A);
	hipMemcpy(h_convolved_signal, d_A, size_A,
		hipMemcpyDeviceToHost);

	for (int i = 0; i < count_A; i++) {
		printf("FFT is %f %f \n", h_convolved_signal[i].x, h_convolved_signal[i].y);
	}

	hipMemcpy(h_C, d_C, count_C * sizeof(double),hipMemcpyDeviceToHost);
	printf("------Count IS: %d - %d = %d \n", count_A, count_B, count_C);
	for (int i = 0; i < count_C+1; i++) {
		printf("------LAD IS: %f \n", h_C[i]);
	}
	*a = h_C[(count_A - count_B) + 1];

	// Allocate host memory for the convolution result
	//Complex* h_convolved_signal_ref = (Complex*)malloc(sizeof(Complex) * size_A);

	// Convolve on the host


	//Destroy CUFFT context
	hipfftDestroy(planA);
	hipfftDestroy(planB);

	// cleanup memory
	free(h_A);
	free(h_B);
	//free(h_padded_signal);
	//free(h_padded_filter_kernel);
	//free(h_convolved_signal_ref);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	// Clean up memory



	return EXIT_SUCCESS;

}



int readFile(int **grades, char *addr) {
	FILE *fp;
	int temp;
	//grades = NULL;
	int count = 1;
	long index;

	


	fp = fopen(addr, "rb+");

	while (fscanf(fp, "%d", &temp) != EOF)

	{


		if (*grades == NULL)

		{

			*grades = (int *)malloc(sizeof(temp));
			**grades = temp;

			printf("The grade is %d\r\n", temp);
		}

		else
		{
			//printf("The grade is realloc %d\r\n", temp);
			count++;
			*grades = (int *)realloc(*grades, sizeof(int)*count);
			index = count - 1;
			(*grades)[index] = temp;
			//printf("the index is %d\r\n",index);

		}

	}
	printf("Done Total %d numbers \n", count);
	fclose(fp);
	temp = 0;
	/*
	while (index >= 0)
	{

	printf("the read value is %d\r\n", (*grades)[temp]);
	index--;
	temp++;

	} */

	return(count);
	
}

void concatenate_string(char *original, char *add)
{
	while (*original)
		original++;

	while (*add)
	{
		*original = *add;
		add++;
		original++;
	}
	*original = '\0';
}

/**
* Program main
*/
int main(int argc, char **argv)
{
	//char **strings1 = (char**)malloc(10 * sizeof(char*));
	char arr1[10][30];
	FILE * database;
	char buffer1[30];
	int Count1 = 0;

	database = fopen("SongNames.txt", "r");

	if (NULL == database)
	{
		perror("opening database");
		return (-1);
	}

	while (EOF != fscanf(database, "%[^\n]\n", buffer1))
	{
		//printf("> %s\n", buffer1);
		strcpy(arr1[Count1], buffer1);
		Count1++;
	}
	fclose(database);

	char arr2[10][30];
	FILE * database2;
	char buffer2[30];
	int Count2 = 0;
	
	database2 = fopen("SampleNames.txt", "r");

	if (NULL == database2)
	{
		perror("opening database");
		return (-1);
	}

	while (EOF != fscanf(database2, "%[^\n]\n", buffer2))
	{
		//printf("> %s\n", buffer2);
		strcpy(arr2[Count2], buffer2);
		Count2++;
	}
	fclose(database2);

	for (int i = 0; i < Count1; i++) {
		for (int j = 0; j < Count2; j++) {
			double a = 0.0;
	
			char path1[30]="songs/";
			char path2[30] = "samples/";
			concatenate_string(path1, arr1[i]);
			concatenate_string(path2, arr2[i]);

			CompareWav(path1, path2,&a);

			printf("%s --- %s --- %s > %s : Similarity rate: %d\n", path1,path2, arr1[i], arr2[j], a);
		}
	}


	
	//printf("------return is: %f \n", a);

	/*

	struct dirent *de;  // Pointer for directory entry 

						// opendir() returns a pointer of DIR type.  
	DIR *dr = opendir(".");

	if (dr == NULL)  // opendir returns NULL if couldn't open directory 
	{
		printf("Could not open current directory");
		return 0;
	}


	// for readdir() 
	while ((de = readdir(dr)) != NULL)
		printf("%s\n", de->d_name);

	closedir(dr);
	*/

}
